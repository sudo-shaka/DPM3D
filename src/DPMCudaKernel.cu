#include "hip/hip_runtime.h"
#include "../include/Cell.hpp"
#include<cmath>
#include <glm/geometric.hpp>
#include<stdio.h>
#include<glm/glm.hpp>
#include<glm/vec3.hpp>
#include<glm/mat3x3.hpp>
#include <glm/gtx/norm.hpp>

__global__ void cuShapeForce2D(float dt,int MaxNV, int NCELLS, cudaDPM::Cell2D *Cells, cudaDPM::Vertex2D* Verts){
  int ci = blockIdx.x;
  int vi = threadIdx.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int indexm = index-1;
  int indexm2 = index-2;
  int indexp = index+1;
  int indexp2 = index+2;
  if(vi == Cells[ci].NV-1){
    indexp -= Cells[ci].NV;
    indexp2 = indexp+1;
  }
  else if(vi == Cells[ci].NV-2){
    indexp2 -= Cells[ci].NV;
  }
  if(vi == 0){
    indexm += (Cells[ci].NV);
    indexm2 = indexm - 1;
  }
  else if(vi == 1){
    indexm2 += (Cells[ci].NV);
  }

  float PartialArea = 0.0, areaStrain = 0.0;

  if(vi < Cells[ci].NV && ci < NCELLS){
    //ForceVars
    float Fxa = 0, Fya = 0, Fxb = 0, Fyb = 0, Fxp =0 ,Fyp=0;
    float Fys = 0, Fxs = 0;

    //PerimeterForceUpdate
    float lvxm,lvx;
    float lvym,lvy;
    float ulvxm,ulvx;
    float ulvym,ulvy;
    float dlim1, dli;
    float length, lengthm;
    float l0 = Cells[ci].l0;
    lvx = Verts[indexp].X - Verts[index].X;
    lvy = Verts[indexp].Y - Verts[index].Y;
    lvxm = Verts[index].X - Verts[indexm].X;
    lvym = Verts[index].Y - Verts[indexm].Y;
    length = sqrt(lvx*lvx + lvy*lvy);
    lengthm = sqrt(lvxm*lvxm + lvym*lvym);
    ulvx = lvx/length;
    ulvy = lvy/length;
    ulvxm = lvxm/lengthm;
    ulvym = lvym/lengthm;
    dli = length/l0 - 1.0;
    dlim1 = lengthm/l0 - 1.0;
    Fxp = Cells[ci].Kl*((sqrt(Cells[ci].a0)/l0))*(dli*ulvx- dlim1*ulvxm);
    Fyp = Cells[ci].Kl*((sqrt(Cells[ci].a0)/l0))*(dli*ulvy- dlim1*ulvym);

    //BendingForceUpdate
    float rho0 = sqrt(Cells[ci].a0);
    float fb = Cells[ci].Kb*(rho0/(l0*l0));
    float six, sixp, sixm;
    float siy, siyp, siym;
    six = lvx - lvxm;
    siy = lvy - lvym;
    sixp = (Verts[indexp2].X - Verts[indexp].X) - lvx;
    siyp = (Verts[indexp2].Y - Verts[indexp].Y) - lvy;
    sixm = lvxm - (Verts[indexm].X - Verts[indexm2].X);
    siym = lvym - (Verts[indexm].Y - Verts[indexm2].Y);
    Fxb = fb*(2.0*six - sixm - sixp);
    Fyb = fb*(2.0*siy - siym - siyp);

    //AreaForceUpdate
    Cells[ci].Area = 0.0;
    PartialArea = 0.5*((Verts[indexm].X + Verts[index].X)*(Verts[indexm].Y - Verts[index].Y));
    atomicAdd(&Cells[ci].Area, PartialArea);
    if(Cells[ci].Area < 0.0){Cells[ci].Area *= -1.0;}
    areaStrain = (Cells[ci].Area/Cells[ci].a0) - 1.0;
    Fxa = (Cells[ci].Ka/(sqrt(Cells[ci].a0)))*0.5*areaStrain*(Verts[indexm].Y-Verts[indexp].Y);
    Fya = (Cells[ci].Ka/(sqrt(Cells[ci].a0)))*0.5*areaStrain*(Verts[indexm].X-Verts[indexp].X);


    //Driving Force Update
    float Fxd=0.0, Fyd=0.0;
    if(Cells[ci].v0 != 0.0){
      float rx,ry,psiVi,v0tmp,rscale,dpsi;
      rx = Verts[index].X - Cells[ci].COMX;
      ry = Verts[index].Y - Cells[ci].COMY;
      psiVi = atan2(rx,ry);
      dpsi = psiVi - Cells[ci].psi;
      dpsi -= 2.0*M_PI*round(dpsi/(2.0*M_PI));
      v0tmp = Cells[ci].v0*exp(-(dpsi*dpsi)/(2.0*Cells[ci].Ds*Cells[ci].Ds)) + Cells[ci].vmin;
      rscale = sqrt(rx*rx + ry*ry);
      Fxd = v0tmp*(rx/rscale);
      Fyd = v0tmp*(ry/rscale);
    }

    //Stick force update
    float dx = Cells[ci].COMX - Verts[index].X;
    float dy = Cells[ci].COMY - Verts[index].Y;
    float norm = sqrt(dx*dx + dy*dy);
    if(Verts[index].Y < 0.0){
      Fys -= Verts[index].Y/l0 * Cells[ci].Ks;
    }
 /*   else if(lvx < 0.0f && Verts[index].Y < l0){
        Fxs -= Cells[ci].Ks * ((1.0f-Verts[index].Y)/(l0)/l0) *
          (Verts[index].X-Cells[ci].COMX)/norm;
        Fys -= Cells[ci].Ks * ((1.0f-Verts[index].Y)/(l0)/l0) *
          (Verts[index].Y-Cells[ci].COMY)/norm;
    }*/

    //Update forces and Positions
    Verts[index].Fx = Fxa+Fxp+Fxb+Fxd+Fys;
    Verts[index].Fy = Fya+Fyp+Fyb+Fyd+Fxs;
    Verts[index].Vx = 0.5*dt*Verts[index].Fx;
    Verts[index].Vy = 0.5*dt*Verts[index].Fy;
    Verts[index].X += dt*Verts[index].Fx;
    Verts[index].Y += dt*Verts[index].Fy;

    __syncthreads();

  }
}


__global__ void cuRetractingForce2D(float dt,int MaxNV, float Kc, float L, int NCELLS, cudaDPM::Cell2D *Cells, cudaDPM::Vertex2D *Verts){

  int ci = blockIdx.x;
  int vi = threadIdx.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int indexm = index-1;
  int indexm2 = index-2;
  int indexp = index+1;
  int indexp2 = index+2;
  if(vi == Cells[ci].NV-1){
    indexp -= Cells[ci].NV;
    indexp2 = indexp+1;
  }
  else if(vi == Cells[ci].NV-2){
    indexp2 -= Cells[ci].NV;
  }
  if(vi == 0){
    indexm += (Cells[ci].NV);
    indexm2 = indexm - 1;
  }
  else if(vi == 1){
    indexm2 += (Cells[ci].NV);
  }
  float rij,xij,ftmp=0.0,dx,dy;

  if(vi < Cells[ci].NV && ci < NCELLS){
    //for all other cells, use crossing test to see if there is an overlap.
    int cj_vj_i;
    int cj_vj_j;
    bool overlaps = false;
    Cells[ci].COMX = 0.0;
    Cells[ci].COMY = 0.0;
    atomicAdd(&Cells[ci].COMX, Verts[index].X);
    atomicAdd(&Cells[ci].COMY, Verts[index].Y);
    Cells[ci].COMX /= Cells[ci].NV;
    Cells[ci].COMY /= Cells[ci].NV;
    int i,j;
    float dxi, dyi,dxj,dyj;

    __syncthreads();
    for(int cj=0;cj<NCELLS;cj++){
      overlaps = false;
      for(i=0,j = Cells[cj].NV-1; i<Cells[cj].NV; j = i++){
        cj_vj_i = (cj*MaxNV)+i;
        cj_vj_j = (cj*MaxNV)+j;
        dxi = Verts[index].X-Verts[cj_vj_i].X;
        dxj = Verts[index].X-Verts[cj_vj_j].X;
        dyi = Verts[index].Y-Verts[cj_vj_i].Y;
        dyj = Verts[index].Y-Verts[cj_vj_j].Y;
        if(abs(dxi) > L || abs(dxj) > L){
          dxi -= L*floor(dxi/L);
          dxj -= L*floor(dxj/L);
        }
        if(abs(dyi) > L || abs(dyj) > L){
          dyi -= L*round(dyi/L);
          dyj -= L*round(dyj/L);
        }

        if(ci != cj){
          if( ((dyi>0) != (dyj>0)) &&
              (0 < (dxj-dxi) * (0-dyi) / (dyj-dyi) + dxi) ){
            overlaps = !overlaps;
          }
        }
      }
      if(overlaps){
        break;
      }
    }

    if(overlaps){
      dx = Cells[ci].COMX - Verts[index].X;
      dy = Cells[ci].COMY - Verts[index].Y;
      rij = abs(sqrt(dx*dx + dy*dy));
      xij = rij/(2*Cells[ci].r0);
      ftmp = Kc*(1-xij);
      Cells[ci].U += 0.5 * Kc * pow(1-xij,2);
      Verts[index].Fx += ftmp * (dx/rij);
      Verts[index].Fy += ftmp * (dy/rij);
      Verts[index].Vx = 0.5*dt*Verts[index].Fx;
      Verts[index].Vy = 0.5*dt*Verts[index].Fy;
      Verts[index].X += dt*(ftmp * (dx/rij));
      Verts[index].Y += dt*(ftmp * (dy/rij));
    }

    __syncthreads();
  }
}

__global__ void cuShapeForce3D(float dt,int NCELLS,cudaDPM::Cell3D* Cells, cudaDPM::Vertex3D *Verts, glm::ivec3 *Triangles){
  int ci = blockIdx.x;
  int fi = threadIdx.x;
  int NV = Cells[0].NV;
  float l0 = Cells[0].l0;
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if(ci < NCELLS && fi < Cells[ci].ntriangles){
    glm::ivec3 FaceIndex = Triangles[index];
    glm::vec3 P0 = {Verts[FaceIndex.x+(ci*NV)].X,Verts[FaceIndex.x+(ci*NV)].Y,Verts[FaceIndex.x+(ci*NV)].Z};
    glm::vec3 P1 = {Verts[FaceIndex.y+(ci*NV)].X,Verts[FaceIndex.y+(ci*NV)].Y,Verts[FaceIndex.y+(ci*NV)].Z};
    glm::vec3 P2 = {Verts[FaceIndex.z+(ci*NV)].X,Verts[FaceIndex.z+(ci*NV)].Y,Verts[FaceIndex.z+(ci*NV)].Z};
    glm::vec3 COM = {Cells[ci].COMX,Cells[ci].COMY,Cells[ci].COMZ};
    glm::mat3 Forces = {0,0,0,0,0,0,0,0,0}, Positions = {P0,P1,P2};
    int j;

      //Volume Force Update
    float VolumeStrain = (Cells[ci].Volume/Cells[ci].v0) - 1.0;
    glm::vec3 A = P1-P0;
    glm::vec3 B = P2-P0;
    glm::vec3 Direction = glm::normalize(glm::cross(A,B));

    for(j=0;j<3;j++){
      Forces[j] -= Cells[ci].Kv * 0.5f * VolumeStrain * Direction;
    }

    //SurfaceArea Force Update

    //Keep equalateral
    glm::mat3 lv,ulv;
    glm::vec3 length,dli;
    lv[0] = Positions[1] - Positions[0];
    lv[1] = Positions[2] - Positions[1];
    lv[2] = Positions[0] - Positions[2];
    length[0] = sqrt(glm::dot((lv[0]),(lv[0])));
    length[1] = sqrt(glm::dot((lv[1]),(lv[1])));
    length[2] = sqrt(glm::dot((lv[2]),(lv[2])));
    ulv[0] = lv[0]/length[0];
    ulv[1] = lv[1]/length[1];
    ulv[2] = lv[2]/length[2];
    dli[0] = length[0]/l0 - 1.0f;
    dli[1] = length[1]/l0 - 1.0f;
    dli[2] = length[2]/l0 - 1.0f;
    Forces[0] += Cells[ci].Ka * (dli[0]*ulv[0]-dli[2]*ulv[2]);
    Forces[1] += Cells[ci].Ka * (dli[1]*ulv[1]-dli[0]*ulv[0]);
    Forces[2] += Cells[ci].Ka * (dli[2]*ulv[2]-dli[1]*ulv[1]);

    //Just keep area constant
/*
    float Area = 0.5 * glm::l2Norm(glm::cross((P1-P0),(P2-P0)));
    float AreaStrain=(Area/Cells[ci].a0) - 1.0;
    glm::vec3 center = (P0+P1+P2)/3.0f;
    Forces[0] += Cells[ci].Ka * 0.5f * AreaStrain * glm::normalize(center-P0);
    Forces[1] += Cells[ci].Ka * 0.5f * AreaStrain * glm::normalize(center-P1);
    Forces[2] += Cells[ci].Ka * 0.5f * AreaStrain * glm::normalize(center-P2);
*/

    //Sticking to surface
    for(j=0;j<3;j++){
      length[j] = Positions[j].z;
      if(Positions[j].z < 0.0){
        Forces[j].z -= Cells[ci].Ks * length[j]/l0;
      }
      else if((A.x*B.y - A.y*B.x) < 0.0f && length[j] < l0){
        Forces[j] += Cells[ci].Ks * ((1.0f-length[j]/(l0))/l0) * glm::normalize(Positions[j]-COM);
      }
    }


    //Update Position and Forces
    Verts[FaceIndex.x+(ci*NV)].Fx = Forces[0][0];
    Verts[FaceIndex.x+(ci*NV)].Fy = Forces[0][1];
    Verts[FaceIndex.x+(ci*NV)].Fz = Forces[0][2];

    Verts[FaceIndex.y+(ci*NV)].Fx = Forces[1][0];
    Verts[FaceIndex.y+(ci*NV)].Fy = Forces[1][1];
    Verts[FaceIndex.y+(ci*NV)].Fz = Forces[1][2];

    Verts[FaceIndex.z+(ci*NV)].Fx = Forces[2][0];
    Verts[FaceIndex.z+(ci*NV)].Fy = Forces[2][1];
    Verts[FaceIndex.z+(ci*NV)].Fz = Forces[2][2];

    Verts[FaceIndex.x+(ci*NV)].X += dt*Verts[FaceIndex.x+(ci*NV)].Fx;
    Verts[FaceIndex.x+(ci*NV)].Y += dt*Verts[FaceIndex.x+(ci*NV)].Fy;
    Verts[FaceIndex.x+(ci*NV)].Z += dt*Verts[FaceIndex.x+(ci*NV)].Fz;

    Verts[FaceIndex.y+(ci*NV)].X += dt*Verts[FaceIndex.y+(ci*NV)].Fx;
    Verts[FaceIndex.y+(ci*NV)].Y += dt*Verts[FaceIndex.y+(ci*NV)].Fy;
    Verts[FaceIndex.y+(ci*NV)].Z += dt*Verts[FaceIndex.y+(ci*NV)].Fz;

    Verts[FaceIndex.z+(ci*NV)].X += dt*Verts[FaceIndex.z+(ci*NV)].Fx;
    Verts[FaceIndex.z+(ci*NV)].Y += dt*Verts[FaceIndex.z+(ci*NV)].Fy;
    Verts[FaceIndex.z+(ci*NV)].Z += dt*Verts[FaceIndex.z+(ci*NV)].Fz;
  }

  __syncthreads();
}


__global__ void cuRepellingForce3D(float dt, int NCELLS, int NT, float L, float Kc,
                                   cudaDPM::Cell3D* Cells, cudaDPM::Vertex3D
                                   *Verts, glm::ivec3 *Triangles){

  int ci = blockIdx.x;
  int fi = threadIdx.x;
  int NV = Cells[0].NV;
  int cj,fj;
  float l0 = Cells[ci].l0;
  float dist;
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if(ci < NCELLS && fi < Cells[ci].ntriangles && Kc > 0.0001){
    glm::vec3 COM = {Cells[ci].COMX,Cells[ci].COMY,Cells[ci].COMZ};
    glm::ivec3 FaceIndexI = Triangles[index], FaceIndexJ;
    glm::mat3 Forces = {0,0,0,0,0,0,0,0,0}, PI, PJ;
    PI[0] = {Verts[FaceIndexI.x+(ci*NV)].X,Verts[FaceIndexI.x+(ci*NV)].Y,Verts[FaceIndexI.x+(ci*NV)].Z};
    PI[1] = {Verts[FaceIndexI.y+(ci*NV)].X,Verts[FaceIndexI.y+(ci*NV)].Y,Verts[FaceIndexI.y+(ci*NV)].Z};
    PI[2] = {Verts[FaceIndexI.z+(ci*NV)].X,Verts[FaceIndexI.z+(ci*NV)].Y,Verts[FaceIndexI.z+(ci*NV)].Z};
    glm::vec3 normali, FaceCenterI, FaceCenterJ, rij;

    FaceCenterI = (PI[0]+PI[1]+PI[2])/3.0f;

    normali = glm::cross((PI[1]-PI[0]),(PI[2]-PI[0]));
    for(cj=0;cj<NCELLS;cj++){
      if(ci != cj){
        for(fj=0;fj<NT;fj++){
          FaceIndexJ = Triangles[fj+(cj*NT)];
          PJ[0] = {Verts[FaceIndexJ.x+(cj*NV)].X,Verts[FaceIndexJ.x+(cj*NV)].Y,Verts[FaceIndexJ.x+(cj*NV)].Z};
          PJ[1] = {Verts[FaceIndexJ.y+(cj*NV)].X,Verts[FaceIndexJ.y+(cj*NV)].Y,Verts[FaceIndexJ.y+(cj*NV)].Z};
          PJ[2] = {Verts[FaceIndexJ.z+(cj*NV)].X,Verts[FaceIndexJ.z+(cj*NV)].Y,Verts[FaceIndexJ.z+(cj*NV)].Z};
          FaceCenterJ = (PJ[0]+PJ[1]+PJ[2])/3.0f;
          rij = FaceCenterJ-FaceCenterI;
          rij -= L*round(rij/L);
          dist = abs(sqrt(glm::dot(rij,rij)));
          if(glm::dot(normali,rij) < 0.0 && dist < l0){
            //Forces[0] += pow(dist,2.0f)*0.5f*Kc*glm::normalize(COM-PI[0]);
            //Forces[1] += pow(dist,2.0f)*0.5f*Kc*glm::normalize(COM-PI[1]);
            //Forces[2] += pow(dist,2.0f)*0.5f*Kc*glm::normalize(COM-PI[2]);
            Forces[0] += (dist/l0)*0.5f*Kc*glm::normalize(COM-PI[0]);
            Forces[1] += (dist/l0)*0.5f*Kc*glm::normalize(COM-PI[1]);
            Forces[2] += (dist/l0)*0.5f*Kc*glm::normalize(COM-PI[2]);
          }
        }
      }
    }
    Verts[FaceIndexI.x+(ci*NV)].Fx = Forces[0][0];
    Verts[FaceIndexI.x+(ci*NV)].Fy = Forces[0][1];
    Verts[FaceIndexI.x+(ci*NV)].Fz = Forces[0][2];

    Verts[FaceIndexI.y+(ci*NV)].Fx = Forces[1][0];
    Verts[FaceIndexI.y+(ci*NV)].Fy = Forces[1][1];
    Verts[FaceIndexI.y+(ci*NV)].Fz = Forces[1][2];

    Verts[FaceIndexI.z+(ci*NV)].Fx = Forces[2][0];
    Verts[FaceIndexI.z+(ci*NV)].Fy = Forces[2][1];
    Verts[FaceIndexI.z+(ci*NV)].Fz = Forces[2][2];

    Verts[FaceIndexI.x+(ci*NV)].X += dt*Verts[FaceIndexI.x+(ci*NV)].Fx;
    Verts[FaceIndexI.x+(ci*NV)].Y += dt*Verts[FaceIndexI.x+(ci*NV)].Fy;
    Verts[FaceIndexI.x+(ci*NV)].Z += dt*Verts[FaceIndexI.x+(ci*NV)].Fz;

    Verts[FaceIndexI.y+(ci*NV)].X += dt*Verts[FaceIndexI.y+(ci*NV)].Fx;
    Verts[FaceIndexI.y+(ci*NV)].Y += dt*Verts[FaceIndexI.y+(ci*NV)].Fy;
    Verts[FaceIndexI.y+(ci*NV)].Z += dt*Verts[FaceIndexI.y+(ci*NV)].Fz;

    Verts[FaceIndexI.z+(ci*NV)].X += dt*Verts[FaceIndexI.z+(ci*NV)].Fx;
    Verts[FaceIndexI.z+(ci*NV)].Y += dt*Verts[FaceIndexI.z+(ci*NV)].Fy;
    Verts[FaceIndexI.z+(ci*NV)].Z += dt*Verts[FaceIndexI.z+(ci*NV)].Fz;
  }

  __syncthreads();

}
