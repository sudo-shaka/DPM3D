#include "hip/hip_runtime.h"
/*
      }
 * =====================================================================================
 *
 *       Filename:  Tissue.cpp
 *
 *    Description:  cudaDPM Tissue interactions and integrators
 *
 *        Version:  1.0
 *        Created:  06/02/2022 09:03:23 AM
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Shaka X,
 *   Organization:  Yale University
 *
 * =====================================================================================
 */

#include"../include/Cell.hpp"
#include"../include/Tissue.hpp"
#include<vector>
#include<thread>
#include<iostream>
#include<cmath>
#include<glm/glm.hpp>
#include<glm/vec3.hpp>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<hip/hip_runtime.h>
#include"../include/DPMCudaKernel.cuh"

namespace cudaDPM{
  Tissue2D::Tissue2D(std::vector<cudaDPM::Cell2D> _Cells, float _phi0){
    phi0 = _phi0;
    Cells = _Cells;
    NCELLS = (int)Cells.size();
    VertDOF = 0;
    MaxNV = 0;
    float sumareas = 0.0;
    for(int ci=0;ci<NCELLS;ci++){
      VertDOF += Cells[ci].NV;
      sumareas += Cells[ci].GetArea();
      if(Cells[ci].NV > MaxNV){
        MaxNV = Cells[ci].NV;
      }
    }
    L = sqrt(sumareas)/_phi0;
    Kc = 1.0;
    U = 0.0;
  }

  void Tissue2D::EulerUpdate(int nsteps, float dt){
    int ci;
    cudaDPM::Vertex2D* VertsCUDA; //pointer of pointers for each cell verticies
    cudaDPM::Cell2D* CellCUDA;

    //Allocate mempory for the data on the CUDA device
    hipError_t m1 = hipMalloc((void **)&VertsCUDA, NCELLS * MaxNV * sizeof(cudaDPM::Vertex2D));
    hipError_t m2 = hipMalloc((void **)&CellCUDA,  NCELLS * sizeof(cudaDPM::Cell2D));
    if(m1 != hipSuccess || m2 != hipSuccess){
      std::cerr << hipGetErrorString(m1) << " : " << hipGetErrorString(m2) << std::endl;
    }

    //For each of the cells copy the vertex data to the memory we stored on the CUDA device
    hipError_t mem;
    for(ci=0;ci<NCELLS;ci++){
      mem = hipMemcpy((VertsCUDA+(ci*MaxNV)),Cells[ci].Verticies.data(),MaxNV * sizeof(cudaDPM::Vertex2D),hipMemcpyHostToDevice);
      if(mem != hipSuccess){
        std::cerr << hipGetErrorString(mem) << std::endl;
        std::cerr << "[!] Error: cannot allocate vertex data to device : ";
      }
    }
    //Copy the cell data to the CUDA device
    mem = hipMemcpy(CellCUDA,Cells.data(),NCELLS * sizeof(cudaDPM::Cell2D),hipMemcpyHostToDevice);
    if(mem != hipSuccess){
      std::cerr << hipGetErrorString(mem) << std::endl;
      std::cerr << "[!] Error: cannot allocate cell data to cudaDevice : ";
    }

    //Start the Kernel
    hipError_t cudaerr;
    for(int step=0;step<nsteps;step++){
      cuShapeForce2D<<<NCELLS,MaxNV>>>(dt,MaxNV,NCELLS,CellCUDA,VertsCUDA);
      cuRetractingForce2D<<<NCELLS,MaxNV>>>(dt,MaxNV,Kc,L,NCELLS,CellCUDA,VertsCUDA);
      cudaerr = hipDeviceSynchronize();
      if(cudaerr != hipSuccess){
        std::cerr << "[!] Error: cannot properly run cudaKernel : ";
        std::cerr << hipGetErrorString(cudaerr) << std::endl;
      }
    }

    //Getting data back
    for(ci=0;ci<NCELLS;ci++){
      mem = hipMemcpy(Cells[ci].Verticies.data(),(VertsCUDA+(ci*MaxNV)), Cells[ci].NV * sizeof(cudaDPM::Vertex2D),hipMemcpyDeviceToHost);
    }
    if(mem != hipSuccess){
      std::cerr << "[!] Error: cannot get data from cuda device : ";
      std::cerr << hipGetErrorString(mem) << std::endl;
    }

    //Freeing up data
    hipFree(CellCUDA);
    hipFree(VertsCUDA);
  }

  void Tissue2D::disperse(){
    std::vector<float> X,Y,Fx,Fy;
    X.resize(NCELLS);Y.resize(NCELLS);
    Fx.resize(NCELLS); Fy.resize(NCELLS);
    float ri,xi,yi,xj,yj,dx,dy,rj,dist;
    float ux,uy,ftmp,fx,fy;
    int i,j, count=0;
    for(i=0;i<NCELLS;i++){
        X[i] = drand48() * L;
        Y[i] = drand48() * L;
    }
    float oldU = 100, dU = 100;
    while(dU > 1e-6){
      U = 0;
      for(i=0;i<NCELLS;i++){
          Fx[i] = 0.0;
          Fy[i] = 0.0;
      }
      for(i=0;i<NCELLS;i++){
          xi = X[i];
          yi = Y[i];
          ri = Cells[i].r0;
          for(j=0;j<NCELLS;j++){
            if(j != i){
              xj = X[j];
              yj = Y[j];
              rj = Cells[j].r0;
              dx = xj-xi;
              dx -= L*round(dx/L);
              dy = yj-yi;
              dy -= L*round(dy/L);
              dist = sqrt(dx*dx + dy*dy);
              if(dist < 0.0) dist *= -1;
              if(dist <= (ri+rj)){
                ux = dx/dist;
                uy = dy/dist;
                ftmp = (1.0-dist/(ri+rj))/(ri+rj);
                fx = ftmp*ux;
                fy = ftmp*uy;
                Fx[i] -= fx;
                Fy[i] -= fy;
                Fy[j] += fy;
                Fx[j] += fx;
                U += 0.5*(1-(dist/(ri+rj))*(1-dist/(ri+rj)));
              }
            }
          }
        }
        for(int i=0; i<NCELLS;i++){
          X[i] += 0.01*Fx[i];
          Y[i] += 0.01*Fy[i];
        }
        dU = U-oldU;
        if(dU < 0.0)
            dU *= -1;
        oldU = U;
        count++;
        if(count > 1e4){
            break;
            std::cout << "Warning: dispersion may not have completed \n";
        }
    }
    for(int i=0;i<NCELLS;i++){
      for(j=0;j<Cells[i].NV;j++){
        Cells[i].Verticies[j].X = Cells[i].r0*(cos(2.0*M_PI*(j+1)/Cells[i].NV)) + X[i];
        Cells[i].Verticies[j].Y = Cells[i].r0*(sin(2.0*M_PI*(j+1)/Cells[i].NV)) + Y[i];
      }
    }
  }

  Tissue3D::Tissue3D(std::vector<cudaDPM::Cell3D> _Cells, float _phi0){
    Cells=_Cells;
    int nv = Cells[0].NV;
    for(cudaDPM::Cell3D c : Cells){
      if((int)c.NV != nv){
        std::cerr << "[!] Error, all cells must have the same number of verticies" << std::endl;
        exit(0);
      }
    }
    phi0 = _phi0;
    NCELLS = Cells.size();
    float volume = 0.0;
    VertDOF = Cells[0].NV * NCELLS;
    TriDOF = Cells[0].ntriangles * NCELLS;
    for(int i=0;i<NCELLS;i++){
      volume += Cells[i].GetVolume();
    }
    L=cbrt(volume)/phi0;
  }

  void Tissue3D::disperse2D(){
    std::vector<float> X,Y,Fx,Fy;
    X.resize(NCELLS);
    Y.resize(NCELLS);
    Fx.resize(NCELLS);
    Fy.resize(NCELLS);
    float ri,rj,yi,yj,xi,xj,dx,dy,dist;
    float ux,uy,ftmp,fx,fy;
    int i,j,count;
    for(i=0;i<NCELLS;i++){
      X[i] = drand48() * L;
      Y[i] = drand48() * L;
    }
    float oldU = 100.0f,dU = 100.0f;
    count = 0;
    while(dU > 1e-6){
      U=0.0f;
      for(i=0;i<NCELLS;i++){
        Fx[i] = 0.0f;
        Fy[i] = 0.0f;
      }
      for(i=0;i<NCELLS;i++){
        xi = X[i];
        yi = Y[i];
        ri = Cells[i].r0;
        for(j=0;j<NCELLS;j++){
            if(j != i){
              xj = X[j];
              yj = Y[j];
              rj = Cells[j].r0;
              dx = xj-xi;
              dx -= L*round(dx/L);
              dy = yj-yi;
              dy -= L*round(dy/L);
              dist = sqrt(dx*dx + dy*dy);
              if(dist < 0.0f)
                  dist *= -1;
              if(dist <= (ri+rj)){
                ux = dx/dist;
                uy = dy/dist;
                ftmp = (1.0f-dist/(ri+rj))/(ri+rj);
                fx = ftmp*ux;
                fy = ftmp*uy;
                Fx[i] -= fx;
                Fy[i] -= fy;
                Fy[j] += fy;
                Fx[j] += fx;
                U += 0.5*(1-(dist/(ri+rj))*(1-dist/(ri+rj)));
              }
            }
          }
        }
      for(int i=0; i<NCELLS;i++){
        X[i] += 0.01f*Fx[i];
        Y[i] += 0.01f*Fy[i];
      }
      dU = U-oldU;
      if(dU < 0.0f)
          dU *= -1.0f;
        oldU = U;
        count++;
        if(count > 1e5){
          std::cerr << "Warning: Max timesteps for dispersion reached"  << std::endl;
          break;
        }
    }
    for(i=0; i<NCELLS; i++){
      Cells[i].UpdateCOM();
      for(j=0;j<Cells[i].NV;j++){
        Cells[i].Verticies[j].X -= Cells[i].COMX;
        Cells[i].Verticies[j].Y -= Cells[i].COMY;
        Cells[i].Verticies[j].X += X[i];
        Cells[i].Verticies[j].Y += Y[i];
      }
    }
  }

  void Tissue3D::disperse3D(){
    std::vector<glm::vec3> centers;
    std::vector<glm::vec3> forces;
    glm::vec3 rij;
    centers.resize(NCELLS);
    forces.resize(NCELLS);
    int i,j,count=0;
    float ftmp;
    for(i=0;i<NCELLS;i++){
      centers[i].x = drand48() * L;
      centers[i].y = drand48() * L;
      centers[i].z = drand48() * L;
    }
    double oldU = 100, dU = 100, U, dist;
    while(dU > 1e-6){
      U = 0;
      for(i=0;i<NCELLS;i++){
        forces[i] = {0,0,0};
      }
      for(i=0;i<NCELLS;i++){
        for(j=0;j<NCELLS;j++){
          if(i!=j){
            rij = centers[j] - centers[i];
            rij -= L*round(rij/L);
            dist = sqrt(glm::dot(rij,rij));
            if(dist < 0.0){
              dist *= -1;
            }
            if(dist < (Cells[i].r0 + Cells[j].r0)){
              ftmp = (1-dist/(Cells[i].r0+Cells[j].r0)/(Cells[i].r0+Cells[j].r0));
              forces[i] -= ftmp*glm::normalize(rij);
              forces[j] += ftmp*glm::normalize(rij);
              U += 0.5*(1-(dist/(Cells[i].r0+Cells[j].r0))*(1-dist/(Cells[i].r0+Cells[j].r0)));
            }
          }
        }
      }
      for(i=0;i<NCELLS;i++){
        centers[i] += 0.01f*forces[i];
      }
      dU = U - oldU;
      if(dU < 0.0){
        dU *=-1;
      }
      oldU = U;
      count++;
      if(count > 1e5){
        std::cerr << "Warning: Max timesteps for dispersion exceeded" << std::endl;
        break;
      }
    }
    for(i=0;i<NCELLS;i++){
      Cells[i].UpdateCOM();
      for(j=0;j<Cells[i].NV;j++){
        Cells[i].Verticies[j].X -= Cells[i].COMX;
        Cells[i].Verticies[j].Y -= Cells[i].COMY;
        Cells[i].Verticies[j].Z -= Cells[i].COMZ;
        Cells[i].Verticies[j].X += centers[i].x;
        Cells[i].Verticies[j].Y += centers[i].y;
        Cells[i].Verticies[j].Z += centers[i].z;
      }
    }
  }

  void Tissue3D::EulerUpdate(int nsteps, float dt){
    int ci,NV = Cells[0].NV, NT = Cells[0].ntriangles;
    std::vector<std::thread> threads;
    if(NT > 1024){
      std::cerr << "[!] Error: Euler update cannot be completed. Greater than 1024 threads...\n";
      return;
    }

    //initializing pointers to reference host memorty for cuda
    cudaDPM::Cell3D* CellsCuda;
    cudaDPM::Vertex3D* VertsCuda;
    glm::ivec3* TriCuda;

    std::vector<hipError_t> errors;
    errors.resize(3);

    //allocating memory for cuda
    errors[0] = hipMalloc((void **)&CellsCuda, NCELLS  * sizeof(cudaDPM::Cell3D));
    errors[1] = hipMalloc((void **)&VertsCuda, VertDOF * sizeof(cudaDPM::Vertex3D));
    errors[2] = hipMalloc((void **)&TriCuda  , TriDOF  * sizeof(glm::ivec3));

    //Checking for errors
    for(auto& error : errors){
      if(error != hipSuccess){
        std::cerr << hipGetErrorString(error) << std::endl;
        exit(0);
      }
    }

    //Give data to cuda
    for(ci = 0; ci<NCELLS; ci++){
      hipMemcpy(TriCuda+(NT*ci),Cells[ci].FaceIndices.data(),NT * sizeof(glm::ivec3),hipMemcpyHostToDevice);
      hipMemcpy(VertsCuda+(NV*ci),Cells[ci].Verticies.data(),NV * sizeof(cudaDPM::Vertex3D),hipMemcpyHostToDevice);
    }

    for(int s=0; s<nsteps;s++){
      //Need to update volume and center for each timestep (calculated on CPU)
      std::vector<std::thread> threads;
      threads.resize(NCELLS);
      for(ci=0;ci<NCELLS;ci++){
        threads[ci] = std::thread(&cudaDPM::Cell3D::UpdateCOM,&this->Cells[ci]);
      }

      for(auto& th : threads){
        th.join();
      }

      for(ci=0;ci<NCELLS;ci++){
        threads[ci] = std::thread(&cudaDPM::Cell3D::UpdateVolume,&this->Cells[ci]);
      }

      for(auto& th : threads){
        th.join();
      }

      hipMemcpy(CellsCuda,Cells.data(),NCELLS * sizeof(cudaDPM::Cell3D), hipMemcpyHostToDevice);
      cuShapeForce3D<<<NCELLS,NT>>>(dt,NCELLS,CellsCuda,VertsCuda,TriCuda);
      cuRepellingForce3D<<<NCELLS,NT>>>(dt,NCELLS,NT,L,Kc,CellsCuda,VertsCuda,TriCuda);
      hipMemcpy(Cells.data(),CellsCuda,NCELLS * sizeof(cudaDPM::Cell3D),hipMemcpyDeviceToHost);
      for(ci = 0; ci<NCELLS; ci++)
        hipMemcpy(Cells[ci].Verticies.data(),VertsCuda+(NV*ci),NV * sizeof(cudaDPM::Vertex3D),hipMemcpyDeviceToHost);
    }

    //Free mem on cuda
    hipFree(CellsCuda); hipFree(VertsCuda); hipFree(TriCuda);
  }
}
